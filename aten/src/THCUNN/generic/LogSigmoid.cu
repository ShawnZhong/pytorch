
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/LogSigmoid.cu"
#else

#include <THCUNN/common.h>

void THNN_(LogSigmoid_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           THCTensor *buffer)
{
  THCUNN_assertSameGPU(state, 2, input, output);
  THCTensor_(resizeAs)(state, output, input);
  THC_pointwiseApply2<scalar_t, scalar_t>(state, output, input, logSigmoid_updateOutput_functor<scalar_t>());
}

#endif
