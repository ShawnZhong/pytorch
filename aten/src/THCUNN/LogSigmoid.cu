#include "hip/hip_runtime.h"
#include <THCUNN/THCUNN.h>
#include <TH/THHalf.h>
#include <THCUNN/THCHalfAutoNumerics.cuh>
#include <THC/THCApply.cuh>

template <typename T>
struct logSigmoid_updateOutput_functor
{
  __device__ void operator()(T *output, const T *input) const {
    const T max = fmaxType(T{0}, -*input);
    const T z = ::exp(-max) + ::exp(-*input -max);
    *output = -(max + static_cast<T>(std::log(z)));
  }
};


template <>
struct logSigmoid_updateOutput_functor<half> {
  __device__ __forceinline__ void operator()(half* output, const half *input) const {
    float in = __half2float(*input);
    float max = fmaxType(0.f, -in);
    float z = ::exp(-max) + ::exp(-in - max);
    *output = __float2half(-(max + std::log(z)));
  }
};

#include <THCUNN/generic/LogSigmoid.cu>
#include <THC/THCGenerateFloatTypes.h>
